#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mem_read_kernel(const float* __restrict__ data, float* __restrict__ sink, size_t N) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (size_t i = tid; i < N; i += blockDim.x * gridDim.x) {
        sum += data[i];
    }
    sink[tid] = sum;  // 防止编译器优化
}

int main() {
    size_t N = (size_t)1 << 28;  // 1 GiB，共 268,435,456 个 float
    size_t bytes = N * sizeof(float);

    float* d_data;
    float* d_sink;

    hipMalloc(&d_data, bytes);
    hipMalloc(&d_sink, sizeof(float) * 1024);  // dummy 输出防优化

    hipMemset(d_data, 1, bytes);

    dim3 block(256);
    dim3 grid(1024);  // 共 256K 个线程

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    mem_read_kernel<<<grid, block>>>(d_data, d_sink, N);
    hipDeviceSynchronize();

    // timed run
    hipEventRecord(start);
    mem_read_kernel<<<grid, block>>>(d_data, d_sink, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    float gb = bytes / (1024.0f * 1024.0f * 1024.0f);
    float bandwidth = gb / (ms / 1000.0f);

    std::cout << "Memory READ bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(d_data);
    hipFree(d_sink);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
