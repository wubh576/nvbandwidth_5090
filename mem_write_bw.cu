#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mem_write_kernel(float* __restrict__ data, size_t N) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t i = tid; i < N; i += blockDim.x * gridDim.x) {
        data[i] = 3.14f;
    }
}

int main() {
    size_t N = (size_t)1 << 28;  // 1 GiB
    size_t bytes = N * sizeof(float);

    float* d_data;
    hipMalloc(&d_data, bytes);

    dim3 block(256);
    dim3 grid(1024);  // 共 256K 线程

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    mem_write_kernel<<<grid, block>>>(d_data, N);
    hipDeviceSynchronize();

    // timed run
    hipEventRecord(start);
    mem_write_kernel<<<grid, block>>>(d_data, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    float gb = bytes / (1024.0f * 1024.0f * 1024.0f);
    float bandwidth = gb / (ms / 1000.0f);

    std::cout << "Memory WRITE bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(d_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
